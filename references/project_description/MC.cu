/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__device__ float Y0d[10];
__device__ float md[10];
__device__ float alphad[10];
__device__ float nu2d[10];
__device__ float rhod[10];
__device__ float Strd[16];


// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

void strikeInterval(float* K, float T) {

		float fidx = T * 12.0f + 1.0f;
		int i = 0;
		float coef = 1.0f;
		float delta;

		while (i < fidx) {
			coef *= (1.02f);
			i++;
		}

		delta = pow(coef, 1.0f / 8.0f);
		K[15] = coef;

		for (i = 1; i < 16; i++) {
			K[15 - i] = K[15 - i + 1] / delta;
		}
	}

// Set the state for each thread
__global__ void init_curand_state_k(hiprandState* state)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(0, idx, 0, &state[idx]);
}

// Monte Carlo simulation kernel
__global__ void MC_k(float dt, float T, int Ntraj, hiprandState* state, float* sum, int* num){

	float t, S, Y, Y0R, price, sumR, sum2R, StrR;
	int idx = blockDim.x * blockIdx.x + threadIdx.x; 
	hiprandState localState = state[idx];
	/*********************************

	Complete with your code here

	**********************************/

	for (int i = 0; i < Ntraj; i++) {
		t = 0.0f;
		S = 1.0f;
		Y = Y0R;
		while(t<T){
			/*********************************

			Complete with your code here

			**********************************/
			t += dt*dt;
		}
		if (S < 12.0f) {
			price = fmaxf(0.0f, S - StrR) / Ntraj;
			sumR += price;
			sum2R += price * price * Ntraj;
		}
	}
	sum[2 * idx] = sumR;
	sum[2 * idx + 1] = sum2R;

	/* Copy state back to global memory */
	state[idx] = localState;
}

int main(void) {

	float Y0[10] = {logf(0.4f), logf(0.35f), logf(0.31f), logf(0.27f), logf(0.23f), 
					logf(0.2f), logf(0.17f), logf(0.14f), logf(0.11f), logf(0.08f)};
	float m[10] = {logf(0.34f), logf(0.3f), logf(0.27f), logf(0.24f), logf(0.21f), 
					logf(0.18f), logf(0.15f), logf(0.12f), logf(0.09f), logf(0.06f)};
	float alpha[10] = { 0.1f, 0.2f, 0.4f, 0.8f, 1.6f, 3.2f, 6.4f, 12.8f, 25.6f, 51.2f };
	float nu2[10] = { 0.6f, 0.7f, 0.8f, 0.9f, 1.0f, 1.1f, 1.2f, 1.3f, 1.4f, 1.5f };
	float rho[10] = {0.95f, 0.75f, 0.55f, 0.35f, 0.15f, -0.15f, -0.35f, -0.55f, -0.75f, -0.95f};
	
	float Tmt[16] = { 1.0f / 12.0f,  2.0f / 12.0f, 3.0f / 12.0f, 4.0f / 12.0f, 5.0f / 12.0f, 6.0f / 12.0f, 7.0f / 12.0f,
					  8.0f / 12.0f, 9.0f / 12.0f, 10.0f / 12.0f, 11.0f / 12.0f, 1.0f, 1.25f, 1.5f, 1.75f, 2.0f };
	float Str[16];


	hipMemcpyToSymbol(HIP_SYMBOL(Y0d), Y0, 10*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(md), m, 10*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(alphad), alpha, 10*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(nu2d), nu2, 10*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rhod), rho, 10*sizeof(float));

	int NTPB = 128 * 4;
	int NB = 625 * 5;
	int Ntraj = 256*512;
	float dt = sqrtf(1.0f/(1000.0f));
	float StrR, mR, alphaR, betaR, rhoR, Y0R, price, error;

	hiprandState* states;
	hipMalloc(&states, NB*NTPB*sizeof(hiprandState));
	init_curand_state_k <<<NB, NTPB>>> (states);
	float *sum;
	int* num;
	hipMallocManaged(&sum, 2*NB*NTPB*sizeof(float));
	hipMallocManaged(&num, NB * NTPB * sizeof(int));

	FILE* fpt;
	int same;

	char strg[30];
	for(int i=0; i<16; i++){
		strikeInterval(Str, Tmt[i]);
		hipMemcpyToSymbol(HIP_SYMBOL(Strd), Str, 16*sizeof(float));
		MC_k<<<NB,NTPB>>>(dt, Tmt[i], Ntraj, states, sum, num);
		hipDeviceSynchronize();
		for(int j=0; j<16; j++){
			StrR = Str[j];
			sprintf(strg, "Tmt%.4fStr%.4f.csv", Tmt[i], StrR);
			fpt = fopen(strg, "w+");
			fprintf(fpt, "alpha, beta, m, rho, Y0, price, 95cI\n");
			for(int k=0; k< 3125*32; k++){
				same = k + j*(3125*32);
				price = sum[2*same];
				error = 1.96f*sqrtf(sum[2*same+1] - (price * price)) / sqrtf((float)Ntraj);
				/*********************************
				
				Complete with your code here
				
				**********************************/
				
				fprintf(fpt, "%f, %f, %f, %f, %f, %f, %f\n", alphaR, betaR, mR, rhoR, Y0R, price, error);
			}
			fclose(fpt);
		}
	}

	hipFree(states);
	hipFree(sum);
	hipFree(num);

	return 0;
}