#include "hip/hip_runtime.h"
#include "NMCH/random/random.hpp"

namespace nmch::random {

    // Set the state for each thread
    template <typename rnd_state>
    __global__ void init_curand_state_k(rnd_state* state) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(0, idx, 0, &state[idx]);
    };
    

    // Explicit instantiation
    template __global__ void init_curand_state_k(hiprandStateXORWOW_t*);
    template __global__ void init_curand_state_k(hiprandStateMRG32k3a_t*);
    template __global__ void init_curand_state_k(hiprandStatePhilox4_32_10_t*);
    //template __global__ void init_curand_state_k(hiprandStateMtgp32_t*);
    

} // namespace nmch::random