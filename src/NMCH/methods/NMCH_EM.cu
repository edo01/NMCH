#include "hip/hip_runtime.h"
/**
 * Ideas for kernel optimization: 
 * - separate the simulation and the reduction and optimize the reduction as much as possible and then maybe merge the two kernels
 * - use the hardware exp instruction and the hardware sqrt instruction
 * - batch random number generation: USING LESS RANDOM NUMBER STATES??
 https://forums.developer.nvidia.com/t/question-about-optimal-hiprand-use/37752
 */
#include "NMCH/methods/NMCH_EM.hpp"

#define testCUDA(error) (nmch::utils::cuda::checkCUDA(error, __FILE__ , __LINE__))

#define SIZE 32

namespace nmch::methods::kernels{
    
    template <typename rnd_state>
    __inline__ __device__
    float gamma_distribution(rnd_state* state, float alpha) 
    {
        float d, c, x, v, u, x2;
        
        // if alpha < 1: // we set alpha = alpha + 1 and we use the fact that gamma_alpha = gamma_{alpha + 1} * U^{1/alpha}
        //
        // 1. setup d=a-1/3, c=1/sqrt(9d)
        // 2. generate v = (1 + cX)^3 with x ~ N(0,1)
        // 3. repeat until v > 0
        // 4. generate U ~ U(0,1)
        // 5. if U < 1 - 0.0331x^4 return d*v (or d*v*U^(1/a) if a < 1)
        // 6. if log(U) < 0.5x^2 + d(1-v + log(v)) return dv (or d*v*U^(1/a) if a < 1)
        // else goto 2

        float C;

        if (alpha < 1.0f) {
            C = powf(hiprand_uniform(state), 1.0f / alpha);  // U^(1/alpha) for alpha < 1
            alpha += 1.0f;  // Increment alpha
        } else {
            C = 1.0f;  // No scaling for alpha >= 1
        }

        // step 1
        d = alpha - 1.0f / 3.0f;
        c = 1.0f / sqrtf(9.0f * d);

        while (true) {
            // step 2
            do{ x = hiprand_normal(state); v = 1.0f + c * x; }while (v <= 0.0f);
            v = v * v * v;
            // step 3
            u = hiprand_uniform(state);
            // step 5 and 6
            x2 = x * x;  // Precompute x^2
            if (u < 1.0f - 0.0331f * x2 * x2 || 
                logf(u) < 0.5f * x2 + d * (1.0f - v + logf(v))) return d * v * C;
        }
    }


    template <typename rnd_state>
    __global__ 
    void EM_k1(float S_0, float v_0, float r, float k, float rho, float theta, float sigma, float dt, 
                            float K, int N, rnd_state* state, float* sum, int n)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        extern __shared__ float A[]; // dynamically allocated shared memory
        // pointers to the shared memory
        float *SR, *VR; 
        SR = A; // stock price reduction 
        VR = SR + blockDim.x; // variance reduction

        // get the local state
        rnd_state localState = state[tid];

        int i;
        int N_p; // poisson 
        float lambda, gamma, Vt_next, m, sigma2;

        // initialization of the variance and the price
        float St = S_0;
        float Vt = v_0;
        float vI = 0.0f; // accumulated variance using the trapezoidal rule
        // initializing constansts
        /* WE CAN TRY __expf(-k * dt) instead of expf(-k * dt) 
        is more efficient because it uses the hardware exp instruction 
        but it is less precise*/

        const float exp_kdt = expf(-k * dt); //expf is very expensive to compute
        const float d = 2.0f * k * theta / (sigma * sigma);
        // this part of lambda is constant, no need to compute it at each iteration
        const float lambda_const = (2 * k * exp_kdt) / (sigma * sigma * (1 - exp_kdt)); 

        /*##############################################
         *                  SIMULATION
         *##############################################*/
        for (i = 0; i < N; ++i) { // advancing in time
            // step 1
            lambda = lambda_const * Vt; 
            N_p = hiprand_poisson(&localState, lambda);
            gamma = gamma_distribution(&localState, d + N_p);
            // a lot of divergence here since the gamma distribution is not equally distributed among threads
            Vt_next = (sigma * sigma * (1.0f - exp_kdt) / (2.0f * k)) * gamma;

            // step 2
            vI += (Vt + Vt_next);//*dt; // dt missing????

            // advance the variance
            Vt = Vt_next;
        }
        vI *= dt*0.5; // only done once for numerical stability
        //Vt = v1;
        //step 3 -  Assuming T = 1
        m       = (1.0f / sigma) * (Vt - v_0 - k * theta + k * vI);
        // step 4 
        m       = -0.5f * vI + rho * m;
        // assume S_0 = 1
        sigma2  = (1.0f - rho * rho) * vI;
        //St
        // what happens if we use the hardware exp instruction?
        // what happens if we change hiprand_normal to hiprand_normal2?
        St      = expf(m + sqrtf(sigma2) * hiprand_normal(&localState));

        /*##############################################
         *                  REDUCTION
         *##############################################*/
        SR[threadIdx.x] = fmaxf(0.0f, St - K)/n;
        VR[threadIdx.x] = Vt/n;

        __syncthreads(); // wait for all threads to finish the computation

        i = blockDim.x/2;
        while(i != 0)
        {
            if(threadIdx.x < i)
            {
                SR[threadIdx.x] += SR[threadIdx.x + i];
                VR[threadIdx.x] += VR[threadIdx.x + i];
            }
            __syncthreads(); // wait for all threads to finish the computation
            i /= 2;
        }

        if(threadIdx.x == 0)
        {
            atomicAdd(sum,      SR[0]);
            atomicAdd(sum + 1,  VR[0]);
        }

        // during the exploaration we need to store the current state in the global memory
        state[idx] = localState;
    }

    __inline__ __device__ float warpReduceSum(float val) {
        for (int offset = 16; offset > 0; offset /= 2) {
            // 0xFFFFFFFF each warp contribute
            // val is the register to be shifted
            // offset is the distance to shift
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }
        return val;
    }

    // Perform block-level reduction of the warp reduced values
    __inline__ __device__ float blockReduceSum(float val) {
        /*if the compute capability is lower than 7.0, we are allocating more shared memory than required 
        because the maximum number of threads per block is 512 instead of 1024*/
        static __shared__ float shared[32]; // Shared memory for one value per warp
        int lane = threadIdx.x % 32;        // Lane index within the warp
        int warpId = threadIdx.x / 32;      // Warp index within the block

        // Perform warp-level reduction
        val = warpReduceSum(val); // WE ARE ASSUMING A NUMBER OF THREADS PER BLOCK WHICH IS A MULTIPLE OF THE WARP(not a strong assumption)

        // Write the reduced value of each warp to shared memory (only the first thread of each warp)
        if (lane == 0) shared[warpId] = val; 

        __syncthreads();

        // Let the first warp reduce all warp results
        /*
            At this point some shared memory may be not used
            This may be caused from 2 reasons:
            - we are using a compute capability lower than 7.0
            - the number of threads per block are not the maximum possible
            In this case we will not use all the first warp but just
            the first blockDim.x/32 threads
        */
        val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0; 
        if (warpId == 0) val = warpReduceSum(val); // no divergence here

        return val;
    }

    template <typename rnd_state>
    __global__ 
    void EM_k2(float S_0, float v_0, float r, float k, float rho, float theta, float sigma, float dt, 
                            float K, int N, rnd_state* state, float* sum, int n)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        // get the local state
        rnd_state localState = state[tid];

        int i;
        int N_p; // poisson 
        float lambda, gamma, Vt_next, m, sigma2;

        // initialization of the variance and the price
        float St = S_0;
        float Vt = v_0;
        float vI = 0.0f; // accumulated variance using the trapezoidal rule
        // initializing constansts
        /* WE CAN TRY __expf(-k * dt) instead of expf(-k * dt) 
        is more efficient because it uses the hardware exp instruction 
        but it is less precise*/

        const float exp_kdt = expf(-k * dt); //expf is very expensive to compute
        const float d = 2.0f * k * theta / (sigma * sigma);
        // this part of lambda is constant, no need to compute it at each iteration
        const float lambda_const = (2 * k * exp_kdt) / (sigma * sigma * (1 - exp_kdt)); 

        /*##############################################
         *                  SIMULATION
         *##############################################*/
        for (i = 0; i < N; ++i) { // advancing in time
            // step 1
            lambda = lambda_const * Vt; 
            N_p = hiprand_poisson(&localState, lambda);
            gamma = gamma_distribution(&localState, d + N_p);
            //__syncwarp();
            // a lot of divergence here since the gamma distribution may not equally distributed among threads
            Vt_next = (sigma * sigma * (1.0f - exp_kdt) / (2.0f * k)) * gamma;

            // step 2
            vI += (Vt + Vt_next);//*dt; // dt missing????

            // advance the variance
            Vt = Vt_next;
        }
        vI *= dt*0.5; // only done once for numerical stability
        //Vt = v1;
        //step 3 -  Assuming T = 1
        m       = (1.0f / sigma) * (Vt - v_0 - k * theta + k * vI);
        // step 4 
        m       = -0.5f * vI + rho * m;
        // assume S_0 = 1
        sigma2  = (1.0f - rho * rho) * vI;
        //St
        // what happens if we use the hardware exp instruction?
        // what happens if we change hiprand_normal to hiprand_normal2?
        St      = expf(m + sqrtf(sigma2) * hiprand_normal(&localState));

        /*##############################################
         *                  REDUCTION
         *##############################################*/

        // Perform block-level reduction
        float partialS, partialV;
        partialS = blockReduceSum(fmaxf(0.0f, St - K)/n);
        partialV = blockReduceSum(Vt/n);

        // Use atomicAdd to accumulate the partial sum of the blocks
        if (threadIdx.x == 0){
            atomicAdd(sum, partialS);
            atomicAdd(sum + 1, partialV);
        }

        // during the exploaration we need to store the current state in the global memory
        state[idx] = localState;
    }
    
    template <typename rnd_state>
    __global__ 
    void EM_k3(float S_0, float v_0, float r, float k, float rho, float theta, float sigma, float dt, 
                            float K, int N, rnd_state* state, float* sum, int n)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        
        // For GPUs with compute capability 8.6 maximum shared memory per thread block is 99 KB.
        // In the worst case it is 64 Bytes per thread * 512 = 32 KB which 
        /**
         * We can't use more than 512 threads otherwise we don't have enough shared memory
         *
         */
        __shared__ rnd_state shared_states[512];       

        // copy the state to the shared memory
        shared_states[threadIdx.x] = state[tid];
        __syncthreads(); // I don't think we need this synchronization

        int i;
        int N_p; // poisson 
        float lambda, gamma, Vt_next, m, sigma2;

        // initialization of the variance and the price
        float St = S_0;
        float Vt = v_0;
        float vI = 0.0f; // accumulated variance using the trapezoidal rule
        // initializing constansts
        /* WE CAN TRY __expf(-k * dt) instead of expf(-k * dt) 
        is more efficient because it uses the hardware exp instruction 
        but it is less precise*/
        const float exp_kdt = expf(-k * dt); //expf is very expensive to compute
        const float d = 2.0f * k * theta / (sigma * sigma);
        // this part of lambda is constant, no need to compute it at each iteration
        const float lambda_const = (2 * k * exp_kdt) / (sigma * sigma * (1 - exp_kdt)); 

        /*##############################################
         *                  SIMULATION
         *##############################################*/
        for (i = 0; i < N; ++i) { // advancing in time
            // step 1
            lambda = lambda_const * Vt; 
            N_p = hiprand_poisson(&shared_states[threadIdx.x], lambda);
            gamma = gamma_distribution(&shared_states[threadIdx.x], d + N_p);
            // a lot of divergence here since the gamma distribution is not equally distributed among threads
            Vt_next = (sigma * sigma * (1.0f - exp_kdt) / (2.0f * k)) * gamma;

            // step 2
            vI += (Vt + Vt_next);//*dt; // dt missing????

            // advance the variance
            Vt = Vt_next;
        }
        vI *= dt*0.5; // only done once for numerical stability
        //Vt = v1;
        //step 3 -  Assuming T = 1
        m       = (1.0f / sigma) * (Vt - v_0 - k * theta + k * vI);
        // step 4 
        m       = -0.5f * vI + rho * m;
        // assume S_0 = 1
        sigma2  = (1.0f - rho * rho) * vI;
        //St
        // what happens if we use the hardware exp instruction?
        // what happens if we change hiprand_normal to hiprand_normal2?
        St      = expf(m + sqrtf(sigma2) * hiprand_normal(&shared_states[threadIdx.x]));

        /*##############################################
         *                  REDUCTION
         *##############################################*/

        // Perform block-level reduction
        float partialS, partialV;
        partialS = blockReduceSum(fmaxf(0.0f, St - K)/n);
        partialV = blockReduceSum(Vt/n);

        // Use atomicAdd to accumulate the partial sum of the blocks
        if (threadIdx.x == 0){
            atomicAdd(sum, partialS);
            atomicAdd(sum + 1, partialV);
        }

        // during the exploaration we need to store the current state in the global memory
        state[idx] = shared_states[threadIdx.x];
    }

} // namespace nmch::methods::kernels

namespace nmch::methods
{

    template <typename rnd_state>
    NMCH_EM_K1<rnd_state>::NMCH_EM_K1(int NTPB, int NB, float T, float S_0, float v_0, float r, float k, float rho, float theta, float sigma, int N):
    NMCH<rnd_state>(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N)
    {
        // each thread will have its own state
        state_numbers = NTPB * NB;
    };

    template <typename rnd_state>
    void NMCH_EM_K1<rnd_state>::init_curand_state(unsigned long long seed)
    {
	    nmch::random::init_curand_state_k<<<this->NB, this->NTPB>>>(states, seed);
    };

    template <typename rnd_state>
    void NMCH_EM_K1<rnd_state>::finalize()
    {
        hipFree(sum);
        hipFree(states);
    };

    template <typename rnd_state>
    void NMCH_EM_K1<rnd_state>::print_stats()
    {   
        float real_price = this->S_0 * nmch::utils::NP((this->r + 0.5 * this->sigma * this->sigma)/this->sigma) -
                                        this->K * expf(-this->r) * nmch::utils::NP((this->r - 0.5 * this->sigma * this->sigma) /
                                        this->sigma);
        //call the print_stats of the base class
        NMCH<rnd_state>::print_stats();
        printf("METHOD: EXACT-METHOD\n");
        printf("The estimated price is equal to %f\n", this->strike_price);
        printf("The estimated variance is equal to %f\n", this->variance);
        printf("The true price %f\n", real_price);
        printf("Relative error committed= %f\n", abs((this->strike_price - real_price)/real_price));
        printf("error associated to a confidence interval of 95%% = %f\n",
            1.96 * sqrt((double)(1.0f / (this->state_numbers - 1)) * (this->state_numbers*this->variance - 
            (this->strike_price * this->strike_price)))/sqrt((double)this->state_numbers));
        printf("Execution time %f ms\n", Tim_exec);
        printf("Initialization time %f ms\n", Tim_init);
    }

    // definition of the base class to avoid compilation errors
    template class NMCH_EM_K1<hiprandStateXORWOW_t>;
    template class NMCH_EM_K1<hiprandStateMRG32k3a_t>;
    template class NMCH_EM_K1<hiprandStatePhilox4_32_10_t>;
    
} // NMCH_EM_K1

namespace nmch::methods
{

    template <typename rnd_state>
    NMCH_EM_K1_MM<rnd_state>::NMCH_EM_K1_MM(int NTPB, int NB, float T, float S_0, float v_0, float r, float k, float rho, float theta, float sigma, int N):
    NMCH_EM_K1<rnd_state>(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N)
    {};

    template <typename rnd_state>
    void NMCH_EM_K1_MM<rnd_state>::init(unsigned long long seed)
    {
        
        hipEvent_t start, stop;			
        hipEventCreate(&start);			
        hipEventCreate(&stop);				
        hipEventRecord(start, 0);	

        // one accumulator for the price and one for the variance
        hipMallocManaged(&(this->sum), 2 * sizeof(float));
        hipMemset(this->sum, 0, 2 * sizeof(float));
        hipMalloc(&(this->states), this->state_numbers * sizeof(rnd_state));
        this->init_curand_state(seed);

        hipEventRecord(stop, 0);			
        hipEventSynchronize(stop);			
        hipEventElapsedTime(&(this->Tim_init), start, stop);					
        hipEventDestroy(start);			
        hipEventDestroy(stop);		
    };

    template <typename rnd_state>
    void
    NMCH_EM_K1_MM<rnd_state>::compute()
    {
        hipEvent_t start, stop;			// GPU timer instructions
        hipEventCreate(&start);			// GPU timer instructions
        hipEventCreate(&stop);				// GPU timer instructions
        hipEventRecord(start, 0);			// GPU timer instructions

        kernels::EM_k1<<<this->NB, this->NTPB, 2 * this->NTPB * sizeof(float)>>>(this->S_0, this->v_0,
                this->r, this->k, this->rho, this->theta, this->sigma, this->dt, this->K, this->N, this->states,
                this->sum, this->state_numbers);

        hipDeviceSynchronize(); // we have to synchronize the device since we remove the memcopy

        hipEventRecord(stop, 0);			// GPU timer instructions
        hipEventSynchronize(stop);			// GPU timer instructions
        hipEventElapsedTime(&(this->Tim_exec),			// GPU timer instructions
            start, stop);					// GPU timer instructions
        hipEventDestroy(start);			// GPU timer instructions
        hipEventDestroy(stop);				// GPU timer instructions

        //hipMemcpy(&(this->result), this->sum, sizeof(float), hipMemcpyDeviceToHost);

        this->strike_price = this->sum[0];
        this->variance = this->sum[1];
    };

    // definition of the base class to avoid compilation errors
    template class NMCH_EM_K1_MM<hiprandStateXORWOW_t>;
    template class NMCH_EM_K1_MM<hiprandStateMRG32k3a_t>;
    template class NMCH_EM_K1_MM<hiprandStatePhilox4_32_10_t>;
    
} // NMCH_EM_K1_MM

namespace nmch::methods
{

    template <typename rnd_state>
    NMCH_EM_K2_MM<rnd_state>::NMCH_EM_K2_MM(int NTPB, int NB, float T, float S_0, float v_0, float r, float k, float rho, float theta, float sigma, int N):
    NMCH_EM_K1_MM<rnd_state>(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N)
    {};


    template <typename rnd_state>
    void
    NMCH_EM_K2_MM<rnd_state>::compute()
    {
        hipEvent_t start, stop;			// GPU timer instructions
        hipEventCreate(&start);			// GPU timer instructions
        hipEventCreate(&stop);				// GPU timer instructions
        hipEventRecord(start, 0);			// GPU timer instructions

        kernels::EM_k2<<<this->NB, this->NTPB>>>(this->S_0, this->v_0,
                this->r, this->k, this->rho, this->theta, this->sigma, this->dt, this->K, this->N, this->states,
                this->sum, this->state_numbers);

        hipDeviceSynchronize(); // we have to synchronize the device since we remove the memcopy

        hipEventRecord(stop, 0);			// GPU timer instructions
        hipEventSynchronize(stop);			// GPU timer instructions
        hipEventElapsedTime(&(this->Tim_exec),			// GPU timer instructions
            start, stop);					// GPU timer instructions
        hipEventDestroy(start);			// GPU timer instructions
        hipEventDestroy(stop);				// GPU timer instructions

        //hipMemcpy(&(this->result), this->sum, sizeof(float), hipMemcpyDeviceToHost);

        this->strike_price = this->sum[0];
        this->variance = this->sum[1];
    };

    // definition of the base class to avoid compilation errors
    template class NMCH_EM_K2_MM<hiprandStateXORWOW_t>;
    template class NMCH_EM_K2_MM<hiprandStateMRG32k3a_t>;
    template class NMCH_EM_K2_MM<hiprandStatePhilox4_32_10_t>;
    
} // NMCH_EM_K2_MM

namespace nmch::methods
{
    template <typename rnd_state>
    NMCH_EM_K3_MM<rnd_state>::NMCH_EM_K3_MM(int NTPB, int NB, float T, float S_0, float v_0, float r, float k, float rho, float theta, float sigma, int N):
    NMCH_EM_K2_MM<rnd_state>(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N)
    {};


    template <typename rnd_state>
    void
    NMCH_EM_K3_MM<rnd_state>::compute()
    {
        hipEvent_t start, stop;			// GPU timer instructions
        hipEventCreate(&start);			// GPU timer instructions
        hipEventCreate(&stop);				// GPU timer instructions
        hipEventRecord(start, 0);			// GPU timer instructions

        kernels::EM_k3<<<this->NB, this->NTPB>>>(this->S_0, this->v_0,
                this->r, this->k, this->rho, this->theta, this->sigma, this->dt, this->K, this->N, this->states,
                this->sum, this->state_numbers);

        hipDeviceSynchronize(); // we have to synchronize the device since we remove the memcopy

        hipEventRecord(stop, 0);			// GPU timer instructions
        hipEventSynchronize(stop);			// GPU timer instructions
        hipEventElapsedTime(&(this->Tim_exec),			// GPU timer instructions
            start, stop);					// GPU timer instructions
        hipEventDestroy(start);			// GPU timer instructions
        hipEventDestroy(stop);				// GPU timer instructions

        //hipMemcpy(&(this->result), this->sum, sizeof(float), hipMemcpyDeviceToHost);

        this->strike_price = this->sum[0];
        this->variance = this->sum[1];
    };

    // definition of the base class to avoid compilation errors
    template class NMCH_EM_K3_MM<hiprandStateXORWOW_t>;
    template class NMCH_EM_K3_MM<hiprandStateMRG32k3a_t>;
    template class NMCH_EM_K3_MM<hiprandStatePhilox4_32_10_t>;
    
} // NMCH_EM_K3_MM