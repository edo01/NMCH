#include "hip/hip_runtime.h"
#include "NMCH/methods/NMCH_fw_euler.hpp"

namespace nmch::methods::kernels{

    template <typename rnd_state>
    __global__ void MC_k2(float S_0, float v_0, float r, float k, float theta, float sigma, float dt, 
                            float K, int N, rnd_state* state, float* sum, int n)
    {

        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        rnd_state localState = state[idx]; // in this way we avoid two different series to be the same
        float2 G1, G2;
        float S = S_0;
        float V = v_0;
        extern __shared__ float A[]; // dynamically allocated in the kernel call
        float *R1s, * R2s; 
        R1s = A;
        R2s = R1s + blockDim.x;
        int i;

        // CHECK ME
        float rho = 0.0f;

        for(i = 0; i<N; i++)
        {
            G1 = hiprand_normal2(&localState);
            G2 = hiprand_normal2(&localState);

            S = S + r * S * dt + sqrtf(V)*S*sqrtf(dt)*(rho*G1.x+sqrtf(1-rho*rho)*G2.x);
            V = V + k*(theta - V)*dt + sigma*sqrtf(V)*sqrtf(dt)*G1.x;
            V = abs(V);            
        }

        R1s[threadIdx.x] = fmaxf(0.0f, S - K)/n;
        R2s[threadIdx.x] = V/n;

        __syncthreads(); // wait for all threads to finish the computation

        i = blockDim.x/2;
        while(i != 0)
        {
            if(threadIdx.x < i)
            {
                R1s[threadIdx.x] += R1s[threadIdx.x + i];
                R2s[threadIdx.x] += R2s[threadIdx.x + i];
            }
            __syncthreads(); // wait for all threads to finish the computation
            i /= 2;
        }

        if(threadIdx.x == 0)
        {
            atomicAdd(sum, R1s[0]);
            atomicAdd(sum +1, R2s[0]);
        }

        // if am doing only one montecarlo simulation
        // I haeve to begin again the sequence
        // state[idx] = localState;
    };

} // namespace nmch::methods::kernels


namespace nmch::methods
{

    template <typename rnd_state>
    NMCH_fw_euler<rnd_state>::NMCH_fw_euler(int NTPB, int NB, float T, float S_0, float v_0, float r, float k, float theta, float sigma, int N):
    NMCH<rnd_state>(NTPB, NB, T, S_0, v_0, r, k, theta, sigma, N)
    {
        // each thread will have its own state
        state_numbers = NTPB * NB;
    };

    template <typename rnd_state>
    void NMCH_fw_euler<rnd_state>::init_curand_state()
    {
	    nmch::random::init_curand_state_k<<<this->NB, this->NTPB>>>(states);
    };

    template <typename rnd_state>
    void NMCH_fw_euler<rnd_state>::init()
    {
        // one accumulator for the price and one for the variance
        hipMallocManaged(&sum, 2 * sizeof(float));
        hipMemset(sum, 0, 2 * sizeof(float));
        hipMallocManaged(&states, state_numbers * sizeof(rnd_state));
        this->init_curand_state();
    };

    template <typename rnd_state>
    void NMCH_fw_euler<rnd_state>::finalize()
    {
        hipFree(sum);
        hipFree(states);
    };

    template <typename rnd_state>
    void
    NMCH_fw_euler<rnd_state>::compute()
    {
        float Tim;
        hipEvent_t start, stop;			// GPU timer instructions
        hipEventCreate(&start);			// GPU timer instructions
        hipEventCreate(&stop);				// GPU timer instructions
        hipEventRecord(start, 0);			// GPU timer instructions

        kernels::MC_k2<<<this->NB, this->NTPB, 2 * this->NTPB * sizeof(float)>>>(this->S_0, this->v_0,
                this->r, this->k, this->theta, this->sigma, this->dt, this->K, this->N, this->states, this->sum, this->state_numbers);

        hipDeviceSynchronize(); //we are using the memcopy after.

        hipEventRecord(stop, 0);			// GPU timer instructions
        hipEventSynchronize(stop);			// GPU timer instructions
        hipEventElapsedTime(&Tim,			// GPU timer instructions
            start, stop);					// GPU timer instructions
        hipEventDestroy(start);			// GPU timer instructions
        hipEventDestroy(stop);				// GPU timer instructions

        //hipMemcpy(&(this->result), this->sum, sizeof(float), hipMemcpyDeviceToHost);

        this->strike_price = this->sum[0];
        this->volatility = this->sum[1];
    };

    template <typename rnd_state>
    void NMCH_fw_euler<rnd_state>::print_stats()
    {
        // for now
        printf("The estimated price is equal to %f\n", this->strike_price);
        printf("The estimated volatility is equal to %f\n", this->volatility);
        printf("error associated to a confidence interval of 95%% = %f\n",
            1.96 * sqrt((double)(1.0f / (this->state_numbers - 1)) * (this->state_numbers*sum[1] - (sum[0] * sum[0])))/sqrt((double)this->state_numbers));
        printf("The true price %f\n", this->S_0 * nmch::utils::NP((this->r + 0.5 * this->sigma * this->sigma)/this->sigma) -
                                        this->K * expf(-this->r) * nmch::utils::NP((this->r - 0.5 * this->sigma * this->sigma) / this->sigma));
        //printf("Execution time %f ms\n", Tim);
    }

    // definition of the base class to avoid compilation errors
    template class NMCH_fw_euler<hiprandStateXORWOW_t>;
    template class NMCH_fw_euler<hiprandStateMRG32k3a_t>;
    template class NMCH_fw_euler<hiprandStatePhilox4_32_10_t>;
    
} // namespace nmch::methods
