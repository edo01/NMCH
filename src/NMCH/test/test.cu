#include <string>
#include "NMCH/methods/NMCH_FE.hpp"
#include "NMCH/methods/NMCH_EM.hpp"

using namespace nmch::methods;

int main(int argc, char **argv)
{
	int NTPB = 1024;
	int NB = 512;
	float T = 1.0f;
	float S_0 = 1.0f;
	float v_0 = 0.1f;
	float r = 0.0f;
	float k = 0.5f;
	float rho = -0.7;
	float theta = 0.1f;
	float sigma = 0.3f;
	int N = 50;
	unsigned long long seed = 1234;
	std::string method = "fe"; // default method

	// Parse command line arguments
	for (int i = 1; i < argc; ++i) {
		if (strcmp(argv[i], "--NTPB") == 0 && i + 1 < argc) {
			NTPB = atoi(argv[++i]);
		} else if (strcmp(argv[i], "--NB") == 0 && i + 1 < argc) {
			NB = atoi(argv[++i]);
		} else if (strcmp(argv[i], "--T") == 0 && i + 1 < argc) {
			T = atof(argv[++i]);
		} else if (strcmp(argv[i], "--S_0") == 0 && i + 1 < argc) {
			S_0 = atof(argv[++i]);
		} else if (strcmp(argv[i], "--v_0") == 0 && i + 1 < argc) {
			v_0 = atof(argv[++i]);
		} else if (strcmp(argv[i], "--r") == 0 && i + 1 < argc) {
			r = atof(argv[++i]);
		} else if (strcmp(argv[i], "--k") == 0 && i + 1 < argc) {
			k = atof(argv[++i]);
		} else if (strcmp(argv[i], "--rho") == 0 && i + 1 < argc) {
			rho = atof(argv[++i]);
		} else if (strcmp(argv[i], "--theta") == 0 && i + 1 < argc) {
			theta = atof(argv[++i]);
		} else if (strcmp(argv[i], "--sigma") == 0 && i + 1 < argc) {
			sigma = atof(argv[++i]);
		} else if (strcmp(argv[i], "--N") == 0 && i + 1 < argc) {
			N = atoi(argv[++i]);
		} else if (strcmp(argv[i], "--seed") == 0 && i + 1 < argc) {
			seed = strtoull(argv[++i], nullptr, 10);
		} else if (strcmp(argv[i], "--method") == 0 && i + 1 < argc) {
			method = argv[++i];
		} else if (strcmp(argv[i], "--help") == 0) {
			printf("Usage: %s [options]\n", argv[0]);
			printf("Options:\n");
			printf("  --NTPB <int>       Number of threads per block (default: 1024)\n");
			printf("  --NB <int>         Number of blocks (default: 512)\n");
			printf("  --T <float>        Time period (default: 1.0)\n");
			printf("  --S_0 <float>      Initial stock price (default: 1.0)\n");
			printf("  --v_0 <float>      Initial volatility (default: 0.1)\n");
			printf("  --r <float>        Risk-free rate (default: 0.0)\n");
			printf("  --k <float>        Mean reversion rate (default: 0.5)\n");
			printf("  --rho <float>      Correlation (default: -0.7)\n");
			printf("  --theta <float>    Long-term volatility (default: 0.1)\n");
			printf("  --sigma <float>    Volatility of volatility (default: 0.3)\n");
			printf("  --N <int>          Number of time steps (default: 50)\n");
			printf("  --seed <ull>       Random seed (default: 1234)\n");
			printf("  --method <string>  Method to use: fe or em (default: fe)\n");
			printf("  --help             Display this help message\n");
			return 0;
		}
	}

	if (method == "fe") {
		NMCH_FE_K1_MM<hiprandStateXORWOW_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		nmch.init(seed);
		nmch.compute();
		nmch.print_stats();
		nmch.finalize();
	} else if (method == "em") {
		NMCH_EM_K1_MM<hiprandStateXORWOW_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		nmch.init(seed);
		nmch.compute();
		nmch.print_stats();
		nmch.finalize();
	} else {
		printf("Unknown method: %s\n", method.c_str());
		return 1;
	}
    return 0;
}