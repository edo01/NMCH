#include <string>
#include "NMCH/methods/NMCH_FE.hpp"
#include "NMCH/methods/NMCH_EM.hpp"

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

/**
- default parameters
- N       = 10000

- NMCH_FE_K1_PgM:  
	Execution time 52.874241 ms
	Initialization time 6.773760 ms
- NMCH_FE_K1_PiM
	Execution time 52.875263 ms
	Initialization time 7.162592 ms
- NMCH_FE_K1_MM
	Execution time 52.882721 ms
	Initialization time 7.224960 ms

from this first analysis, it is clear that the use of different memory spaces does not affect the performance
of the code so we shouldn't push in this direction. For semplicity we will then use Memory Management for 
the rest of the project.

This is justified by the fact that the communication between CPU and GPU is not significant, since we are 
moving only two floats.
 */

/**
 * reduction data with 1024*100.000 = 102.400.000 threads
 * using normal reduction we have 4.533248 ms while using warp reduction 
 * we have 2.750464 ms. 
 * while using 1.024.000.000 threads we have 42.272766 ms and 24.312481 ms
 * respectively. 
 */

/**
 * using hiprand_normal4 in FE allows to have 72.066048 ms against the normal version always using 
 * philox4_32_10 which has 85.052193 ms and the normal 53.237823 ms using xorwow.
 */

/**
	presentation ideas: class hierarchy and speedup obtained with each strategy and why we chose a specific
	path.
 */
using namespace nmch::methods;

int main(int argc, char **argv)
{

	int NTPB = 512;
	int NB = 512;
	float T = 1.0f;
	float S_0 = 1.0f;
	float v_0 = 0.1f;
	float r = 0.0f;
	float k = 0.5f;
	float rho = -0.7;
	float theta = 0.1f;
	float sigma = 0.3f;
	int N = 1000;
	unsigned long long seed = 1234;
	std::string method = "fe"; // default method

	// Parse command line arguments
	for (int i = 1; i < argc; ++i) {
		if (strcmp(argv[i], "--NTPB") == 0 && i + 1 < argc) {
			NTPB = atoi(argv[++i]);
		} else if (strcmp(argv[i], "--NB") == 0 && i + 1 < argc) {
			NB = atoi(argv[++i]);
		} else if (strcmp(argv[i], "--T") == 0 && i + 1 < argc) {
			T = atof(argv[++i]);
		} else if (strcmp(argv[i], "--S_0") == 0 && i + 1 < argc) {
			S_0 = atof(argv[++i]);
		} else if (strcmp(argv[i], "--v_0") == 0 && i + 1 < argc) {
			v_0 = atof(argv[++i]);
		} else if (strcmp(argv[i], "--r") == 0 && i + 1 < argc) {
			r = atof(argv[++i]);
		} else if (strcmp(argv[i], "--k") == 0 && i + 1 < argc) {
			k = atof(argv[++i]);
		} else if (strcmp(argv[i], "--rho") == 0 && i + 1 < argc) {
			rho = atof(argv[++i]);
		} else if (strcmp(argv[i], "--theta") == 0 && i + 1 < argc) {
			theta = atof(argv[++i]);
		} else if (strcmp(argv[i], "--sigma") == 0 && i + 1 < argc) {
			sigma = atof(argv[++i]);
		} else if (strcmp(argv[i], "--N") == 0 && i + 1 < argc) {
			N = atoi(argv[++i]);
		} else if (strcmp(argv[i], "--seed") == 0 && i + 1 < argc) {
			seed = strtoull(argv[++i], nullptr, 10);
		} else if (strcmp(argv[i], "--method") == 0 && i + 1 < argc) {
			method = argv[++i];
		} else if (strcmp(argv[i], "--help") == 0) {
			printf("Usage: %s [options]\n", argv[0]);
			printf("Options:\n");
			printf("  --NTPB <int>       Number of threads per block (default: 1024)\n");
			printf("  --NB <int>         Number of blocks (default: 512)\n");
			printf("  --T <float>        Time period (default: 1.0)\n");
			printf("  --S_0 <float>      Initial stock price (default: 1.0)\n");
			printf("  --v_0 <float>      Initial volatility (default: 0.1)\n");
			printf("  --r <float>        Risk-free rate (default: 0.0)\n");
			printf("  --k <float>        Mean reversion rate (default: 0.5)\n");
			printf("  --rho <float>      Correlation (default: -0.7)\n");
			printf("  --theta <float>    Long-term volatility (default: 0.1)\n");
			printf("  --sigma <float>    Volatility of volatility (default: 0.3)\n");
			printf("  --N <int>          Number of time steps (default: 50)\n");
			printf("  --seed <ull>       Random seed (default: 1234)\n");
			printf("  --method <string>  Method to use: fe or em (default: fe)\n");
			printf("  --help             Display this help message\n");
			return 0;
		}
	}

	if (method == "fe") {
		//NMCH_FE_K1_MM<hiprandStateXORWOW_t> nmch1(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		//NMCH_FE_K2_MM<hiprandStateXORWOW_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		//NMCH_FE_K2_PHILOX_MM nmch2P(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		NMCH_FE_K3_MM<hiprandStatePhilox4_32_10_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);

		nmch.init(seed);
		nmch.compute();
		nmch.print_stats();
		nmch.finalize();
		
	} else if (method == "em") {
		//NMCH_EM_K1_MM<hiprandStateXORWOW_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		//NMCH_EM_K2_MM<hiprandStateXORWOW_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);

		NMCH_EM_K3_MM<hiprandStatePhilox4_32_10_t> nmch(NTPB, NB, T, S_0, v_0, r, k, rho, theta, sigma, N);
		nmch.init(seed);
		nmch.compute();
		nmch.print_stats();
		nmch.finalize();
	} else {
		printf("Unknown method: %s\n", method.c_str());
		return 1;
	}
    return 0;
}