#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>

const float K = 1.0f;     
const float S0 = 1.0f;    // the spot values
const float v0 = 0.1f;  
const float r = 0.0f;     // the risk-free interest rate
const float kappa = 0.5f; // the mean reversion rate of the volatility
const float theta = 0.1f; // the long-term volatility
const float sigma = 0.3f; // the volatility of volatility
const float rho = -0.7f;  
const int T = 1;          
const int steps = 1000;   
const float dt = 1.0f / steps; 
const int simulations = 100000; 

// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

/*One-Dimensional Normal Law. Cumulative distribution function. */
double NP(double x) {
	const double p = 0.2316419;
	const double b1 = 0.319381530;
	const double b2 = -0.356563782;
	const double b3 = 1.781477937;
	const double b4 = -1.821255978;
	const double b5 = 1.330274429;
	const double one_over_twopi = 0.39894228;
	double t;

	if (x >= 0.0) {
		t = 1.0 / (1.0 + p * x);
		return (1.0 - one_over_twopi * exp(-x * x / 2.0) * t * (t * (t *
			(t * (t * b5 + b4) + b3) + b2) + b1));
	}
	else {/* x < 0 */
		t = 1.0 / (1.0 - p * x);
		return (one_over_twopi * exp(-x * x / 2.0) * t * (t * (t * (t *
			(t * b5 + b4) + b3) + b2) + b1));
	}
}

// Set the state for each thread
__global__ void init_curand_state(hiprandState_t* state)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	/* Each thread gets same seed, a different sequence
	   number, no offset */
	hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void hestonMonteCarlo(float *d_results, int steps, float dt, float kappa, float theta, float sigma, float rho, hiprandState_t* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //Initialize the random number generator
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState localState = state[idx];

    float St = S0;
    float vt = v0;

    //Simulation time step
    for (int i = 0; i < steps; ++i) {
        float G1 = hiprand_normal(&localState);
        float G2 = hiprand_normal(&localState);

        // Calculate the delta of asset price and volatility
        float dSt = r * St * dt + sqrtf(vt) * St * sqrtf(dt) * (rho * G1 + sqrtf(1 - rho * rho) * G2);
        float dvt = kappa * (theta - vt) * dt + sigma * sqrtf(vt) * sqrtf(dt) * G1;

        St += dSt;
        vt = fabs(vt + dvt); // the function g is either taken to be equal to (·)+ or to | · |
    }
    // E[f(ST )] = E[(S1 − 1)+].
    d_results[tid] = fmaxf(St - K, 0.0f);
}


//Gamma_distribution
__device__ float gamma_distribution(float alpha, hiprandState* state) {
    if (alpha >= 1.0f) {
        float d = alpha - 1.0f / 3.0f;
        float c = 1.0f / sqrtf(9.0f * d);
        float x, v, u;

        while (true) {
            x = hiprand_normal(state);
            v = 1.0f + c * x;
            v = v * v * v;
            if (v <= 0.0f) continue;
            u = hiprand_uniform(state);
            if (u < 1.0f - 0.0331f * (x * x) * (x * x)) return d * v;
            if (logf(u) < 0.5f * x * x + d * (1.0f - v + logf(v))) return d * v;
        }
    } else {
        float u = hiprand_uniform(state);
        return gamma_distribution2(alpha + 1.0f, state) * powf(u, 1.0f / alpha);
    }
}

__global__ void exact_heston(float* d_results_exact, int steps, float dt, hiprandState_t* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = state[tid];

    float St = S0;
    float vt = v0;
    float vI = 0.0f;
    float v1 = 0.0f;

    for (int i = 0; i < steps; ++i) {
        float d = 2.0f * kappa * theta / (sigma * sigma);
        float lambda = (2 * kappa * expf(-kappa * dt) * vt) / (sigma * sigma * (1 - expf(-kappa * dt)));
        int N = hiprand_poisson(&localState, lambda);
        float gamma_sample = gamma_distribution(d + N, &localState);
        
        float vt_next = (sigma * sigma * (1.0f - expf(-kappa * dt)) / (2.0f * kappa)) * gamma_sample;

        vI += 0.5f * (vt + vt_next);

        vt = vt_next;

        if (i == 1) v1 = vt;  
        
    }

    float integral_W = (1.0f / sigma) * (v1 - v0 - kappa * theta + kappa * vI);
    float m = -0.5f * vI + rho * integral_W;
    float sigma2 = (1.0f - rho * rho) * vI;
    float exponent = m + sqrtf(sigma2) * hiprand_normal(&localState);
    St = expf(exponent);
    d_results_exact[tid] = fmaxf(St - K, 0.0f);
}

int main() {
    int NTPB = 256;
    int NB = (simulations + NTPB - 1) / NTPB;

    float *d_results, *d_results_exact;
    hipMalloc((void **)&d_results, simulations * sizeof(float));
    hipMalloc((void **)&d_results_exact, simulations * sizeof(float));

    hiprandState_t* state;
    hipMalloc(&state, simulations * sizeof(hiprandState_t));
    init_curand_state<<<NB, NTPB>>>(state, time(NULL));

    hipEvent_t start, stop;
    float elapsedTime;

    // Euler Discretization
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    euler_heston<<<NB, NTPB>>>(d_results, steps, dt, state);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Euler simulation time: %f ms\n", elapsedTime);

    // Exact Simulation
    hipEventRecord(start, 0);

    exact_heston<<<NB, NTPB>>>(d_results_exact, steps, dt, state);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Exact simulation time: %f ms\n", elapsedTime);

    // Calculate option price for both methods
    float *h_results = (float *)malloc(simulations * sizeof(float));
    float *h_results_exact = (float *)malloc(simulations * sizeof(float));
    hipMemcpy(h_results, d_results, simulations * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_results_exact, d_results_exact, simulations * sizeof(float), hipMemcpyDeviceToHost);

    float option_price_euler = 0.0f;
    float option_price_exact = 0.0f;
    for (int i = 0; i < simulations; ++i) {
        option_price_euler += h_results[i];
        option_price_exact += h_results_exact[i];
    }

    option_price_euler /= simulations;
    option_price_exact /= simulations;
    option_price_euler *= expf(-r);
    option_price_exact *= expf(-r);

    printf("Euler estimated price: %f\n", option_price_euler);
    printf("Exact estimated price: %f\n", option_price_exact);

    free(h_results);
    free(h_results_exact);
    hipFree(d_results);
    hipFree(d_results_exact);
    hipFree(state);

    return 0;
}